
#include <hip/hip_runtime.h>
#include <hipfft/hipfftw.h>
#include <iostream>

using namespace std;

using cutype = hipfftDoubleComplex;

ostream &operator<<(ostream &os, const cutype &rhs) {
    os << "(" << rhs.x << ", " << rhs.y << ")";
    return os;
}

